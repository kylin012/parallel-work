#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "hip/hip_runtime_api.h"

#include<fstream>
#include<sstream>

const int col = 254, elinenum = 53; //����������Ԫ����
const int bytenum = (col - 1) / 32 + 1;   //ÿ��ʵ���е�byte��������

class bitmatrix {
public:
	int mycol;    //����
	int *mybyte;
	bitmatrix() {    //��ʼ��
		mycol = -1;
		mybyte = new int[bytenum];
		for (int i = 0; i < bytenum; i++)
			mybyte[i] = 0;
	}
	void insert(int x) { //���ݶ���
		if (mycol == -1)mycol = x;
		int a = x / 32, b = x % 32;
		mybyte[a] |= (1 << b);
	}
};


bitmatrix *eliminer = new bitmatrix[col], *eline = new bitmatrix[elinenum];
void readdata() {
	using namespace std;
	ifstream ifs;
	ifs.open("D:\\VS��Ŀ\\cuda\\cuda\\eliminer1.txt");  //��Ԫ��
	string temp;
	while (getline(ifs, temp)) {
		istringstream ss(temp);
		int x;
		int trow = 0;
		while (ss >> x) {
			if (!trow)trow = x;    //��һ������Ԫ�ش����к�
			eliminer[trow].insert(x);
		}
	}
	ifs.close();
	ifstream ifs2;
	ifs2.open("D:\\VS��Ŀ\\cuda\\cuda\\eline1.txt");     //����Ԫ��,���뷽ʽ����Ԫ�Ӳ�ͬ
	int trow = 0;
	while (getline(ifs2, temp)) {
		istringstream ss(temp);
		int x;
		while (ss >> x) {
			eline[trow].insert(x);
		}
		trow++;
	}
	ifs2.close();
}
/*void dowork() {  //������Ԫ--����Ԫ��->��Ԫ��
	for (int i = 0; i < elinenum; i++) {
		while (!eline[i].isnull()) {  //ֻҪ����Ԫ�зǿգ�ѭ������
			int tcol = eline[i].mycol;  //����Ԫ�е�����
			if (!eliminer[tcol].isnull())    //������ڶ�Ӧ��Ԫ��
				eline[i].doxor(eliminer[tcol]);
			else {
				eliminer[tcol] = eline[i];    //���ڱ���Ԫ������Ϊ��Ԫ�Ӻ󲻲��������������ֱ����=��ǳ����
				break;
			}
		}
	}
}*/

__global__ void dowork1(int **gelinebyte,int *gelinecol,int **geliminerbyte,int *geliminercol,int i,int elinenum1,int bytenum1) {  //������Ԫ--��Ԫ��->����Ԫ��
	if (geliminercol[i] == -1)return;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	while (bid < elinenum1) {
		if (gelinecol[bid] == i) {
			int temp = tid;
			while (temp < bytenum1) {
				gelinebyte[bid][temp] ^= geliminerbyte[i][temp];
				temp += blockDim.x;
			}
			__syncthreads();
			if (tid == 0) {
				bool f = 0;
				for (int k = bytenum1 - 1; k >= 0&&!f; k--)
					for (int j = 31; j >= 0&&!f; j--)
						if ((gelinebyte[bid][k] & (1 << j)) != 0) {
							gelinecol[bid] = k * 32 + j;
							f = 1;
						}
				if(!f)gelinecol[bid] = -1;
			}
		}
		bid += gridDim.x;
	}
}
__global__ void dowork2(int **gelinebyte, int *gelinecol, int **geliminerbyte, int *geliminercol, int i, int elinenum1) {
	if (blockIdx.x==0&&threadIdx.x == 0) {
		for (int j = 0; j < elinenum1; j++)
			if (gelinecol[j] == i) {
				geliminerbyte[i] = gelinebyte[j];
				geliminercol[i] = gelinecol[j];
				return;
			}
	}
}

void printres(int** celinebyte,int *celinecol) { //��ӡ���
	for (int i = 0; i < elinenum; i++) {
		if (celinecol[i]==-1) { puts(""); continue; }   //���е��������
		for (int j = bytenum - 1; j >= 0; j--) {
			for (int k = 31; k >= 0; k--)
				if ((celinebyte[i][j] & (1 << k)) != 0) {     //һ��������˰�Сʱ�����ǵ���λΪ1ʱ>>�����ڳ�����
					printf("%d ", j * 32 + k);
				}
		}
		puts("");
	}
}
int main() {
	readdata();
	int** geliminerbyte;
	int* geliminercol;
	int** gelinebyte;
	int* gelinecol;
	int** celinebyte = new int*[elinenum];
	int *celinecol=new int[elinenum];
	int** celiminerbyte = new int*[col];
	int *celiminercol=new int[col];
	for (int i = 0; i < elinenum; i++) {
		int* host_1d = new int[bytenum];
		for (int j = 0; j < bytenum; j++)
			host_1d[j] = eline[i].mybyte[j];
		celinecol[i] = eline[i].mycol;
		int* dev_1d;
		hipMalloc((void**)&dev_1d, sizeof(int)*bytenum);// ��ʱ dev_ld ָ�� һƬ�Դ�ռ�
		hipMemcpy(dev_1d, host_1d, sizeof(int)*bytenum, hipMemcpyHostToDevice);
		celinebyte[i] = dev_1d;
	}
	hipMalloc((void**)&gelinebyte, sizeof(int*)*elinenum); // ����һ���Դ��ϵĶ���ָ��
	hipMemcpy(gelinebyte, celinebyte, sizeof(int*)*elinenum, hipMemcpyHostToDevice);
	hipMalloc(&gelinecol, sizeof(int)*elinenum);
	hipMemcpy(gelinecol, celinecol, sizeof(int)*elinenum, hipMemcpyHostToDevice);
	
	for (int i = 0; i < col; i++) {
		int* host_1d = new int[bytenum];
		for (int j = 0; j < bytenum; j++)
			host_1d[j] = eliminer[i].mybyte[j];
		celiminercol[i] = eliminer[i].mycol;
		int* dev_1d;
		hipMalloc((void**)&dev_1d, sizeof(int)*bytenum);// ��ʱ dev_ld ָ�� һƬ�Դ�ռ�
		hipMemcpy(dev_1d, host_1d, sizeof(int)*bytenum, hipMemcpyHostToDevice);
		celiminerbyte[i] = dev_1d;
	}
	hipMalloc((void**)&geliminerbyte, sizeof(int*)*col); // ����һ���Դ��ϵĶ���ָ��
	hipMemcpy(geliminerbyte, celiminerbyte, sizeof(int*)*col, hipMemcpyHostToDevice);
	hipMalloc(&geliminercol, sizeof(int)*col);
	hipMemcpy(geliminercol, celiminercol, sizeof(int)*col, hipMemcpyHostToDevice);
	hipEvent_t start, stop;//��ʱ��
	float etime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//��ʼ��ʱ
	for (int i = col - 1; i >= 0; i--) {
		if (eliminer[i].mycol == -1)
			dowork2 << <1, 1>> > (gelinebyte,gelinecol,geliminerbyte,geliminercol, i, elinenum);
		hipDeviceSynchronize();
		dowork1 << <1024, 1024 >> > (gelinebyte, gelinecol,geliminerbyte,geliminercol, i, elinenum, bytenum);
		hipDeviceSynchronize();
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//ֹͣ��ʱ
	hipEventElapsedTime(&etime, start, stop);
	printf("GPU_LU:%f ms\n", etime);

	//hipMemcpy(celinebyte, gelinebyte, sizeof(int*)*elinenum, hipMemcpyDeviceToHost);
	hipMemcpy(celinecol, gelinecol, sizeof(int)*elinenum, hipMemcpyDeviceToHost);
	//for (int i = 0; i < elinenum; i++)
	//	printf("%d\n", celinecol[i]);
	//printres(celinebyte,celinecol);
	hipFree((void*)gelinebyte);
	hipFree(gelinecol);
	hipFree((void*)geliminerbyte);
	hipFree(geliminercol);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}