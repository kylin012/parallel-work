#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include<stdlib.h>
const int n = 1000;
float m1[n*n];
void init(int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++)
			m1[i*n + j] = 0;
		m1[i*n + i] = 1.0;
		for (int j = i + 1; j < n; j++)
			m1[i*n + j] = rand() % 1000 + 1;
	}
	for (int k = 0; k < n; k++)
		for (int i = k + 1; i < n; i++)
			for (int j = 0; j < n; j++)
				m1[i*n + j] = int((m1[i*n + j] + m1[k*n + j])) % 1000 + 1.0;
}
__global__ void division_kernel(float* data, int k, int N) {	//�����˺���
	int tid = blockDim.x * blockIdx.x + threadIdx.x;//�����߳�����
	if (tid > k&&tid < n)
		data[k*N + tid] /= data[k*N + k];
	return;
}
__global__ void eliminate_kernel(float* data, int k, int N) {	//��ȥ�˺���
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	if (tx == 0)  data[k*N + k] = 1.0;//�Խ���Ԫ����Ϊ 1 
	int row = k + 1 + blockIdx.x;//ÿ���鸺��һ��
	while (row > k&&row < N) {
		int tid = threadIdx.x;
		while (k + 1 + tid < N) {
			int col = k + 1 + tid;
			float temp_1 = data[(row*N) + col];
			float temp_2 = data[(row*N) + k];
			float temp_3 = data[k*N + col];
			data[(row*N) + col] = temp_1 - temp_2 * temp_3;
			tid = tid + blockDim.x;
		}
		__syncthreads();//����ͬ��
		if (threadIdx.x == 0) {
			data[row * N + k] = 0;
		}
		row += gridDim.x;
	}
	return;
}

int main() {
	init(n);
	float* gdata;
	int size = n * n * sizeof(float);
	hipError_t ret;

	ret = hipMalloc(&gdata, size);	//����gpu�ռ�
	if (ret != hipSuccess) {
		printf("hipMalloc gpudata failed!\n");
	}

	ret = hipMemcpy(gdata, m1, size, hipMemcpyHostToDevice);//�����ݴ����� GPU ��

	if (ret != hipSuccess) {
		printf("hipMemcpyHostToDevice failed!\n");
	}
	dim3 grid(1024, 1);//�߳̿�
	dim3 block(1024, 1);//�߳�����
	hipEvent_t start, stop;//��ʱ��
	float etime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);//��ʼ��ʱ

	for (int k = 0; k < n; k++) {
		division_kernel << <grid, block >> > (gdata, k, n);//�����������ĺ˺���
		hipDeviceSynchronize();//CPU �� GPU ֮���ͬ������
		ret = hipGetLastError();
		if (ret != hipSuccess) {
			printf("division_kernel failed, %s\n", hipGetErrorString(ret));
		}
		eliminate_kernel << <grid, block >> > (gdata, k, n);//������ȥ����ĺ˺���
		hipDeviceSynchronize();
		ret = hipGetLastError();
		if (ret != hipSuccess) {
			printf("eliminate_kernel failed, %s\n", hipGetErrorString(ret));
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//ֹͣ��ʱ
	hipEventElapsedTime(&etime, start, stop);
	printf("GPU_LU:%f ms\n", etime);

	ret = hipMemcpy(m1, gdata, size, hipMemcpyDeviceToHost);//�����ݴ��� CPU ��
	/*for(int i=0;i<n;i++){
		for (int j = 0; j < n; j++)
			printf("%.2f ", m1[i*n+j]);
		puts("");
	}*/
	if (ret != hipSuccess) {
		printf("hipMemcpyDeviceToHost failed!\n");
	}
	hipFree(gdata);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}