#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
   int gridnum=blockDim.x*gridDim.x;
    for(;tid<N;tid+=gridnum)
        c[tid] = 2 * a[tid] + b[tid];
}

int main()
{
    int *a, *b, *c;
    int deviceId;
    int numberofSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberofSMs,hipDeviceAttributeMultiprocessorCount, deviceId);

    size_t size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

     // Initialize memory
  for( int i = 0; i < N; ++i )
  {
    a[i] = 2;
    b[i] = 1;
    c[i] = 0;
  }
    

    int threads_per_block = 1024;
    int number_of_blocks = numberofSMs*32;

   hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);
     // Initialize memory
     
     
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipDeviceSynchronize();

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
