
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
 int i=threadIdx.x+blockIdx.x*blockDim.x;
 int gridnum=blockDim.x*gridDim.x;
  for(; i < N; i+=gridnum)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a,size);
  hipMallocManaged(&b,size);
  hipMallocManaged(&c,size);
  
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<1,1024>>>(c, a, b, N);
 
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());
  
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
